/*
https://blog.csdn.net/weixin_33708432/article/details/86365370
nvcc test.cu -lcublas -w -O2
*/

#include "hip/hip_runtime.h"
#include "hipblas.h"

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <ctime>

using namespace std;

int main()
{
    srand(time(0));
    int M = 2;            //矩阵A的行，矩阵C的行
    int N = 3;            //矩阵A的列，矩阵B的行
    int K = 4;            //矩阵B的列，矩阵C的列

    float *h_A = (float*)malloc(sizeof(float)*M*N);
    float *h_B = (float*)malloc(sizeof(float)*N*K);
    float *h_C = (float*)malloc(sizeof(float)*M*K);

    for (int i = 0; i < M*N; i++)
    {
        h_A[i] = rand() % 10;
        cout << h_A[i] << "  ";
        if ((i + 1) % N == 0)
            cout << endl;        
    }
    cout << endl;

    for (int i = 0; i < N*K; i++)
    {
        h_B[i] = rand() % 10;
        cout << h_B[i] << "  ";
        if ((i + 1) % K == 0)
            cout << endl;
    }
    cout << endl;

    float *d_A, *d_B, *d_C,*d_CT;
    hipMalloc((void**)&d_A, sizeof(float)*M*N);
    hipMalloc((void**)&d_B, sizeof(float)*N*K);
    hipMalloc((void**)&d_C, sizeof(float)*M*K);

    hipMemcpy(d_A, h_A, M*N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N*K * sizeof(float), hipMemcpyHostToDevice);

    float alpha = 1;
    float beta = 0;

    //C=A*B
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSgemm(handle,
        HIPBLAS_OP_N,  
        HIPBLAS_OP_N,   
        K,                    //矩阵B的列数
        M,                    //矩阵A的行数
        N,                    //矩阵A的列数
        &alpha,           
        d_B,            
        K,                    
        d_A,         
        N,         
        &beta,          
        d_C,           
        K);

    hipMemcpy(h_C, d_C, M*K * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < M*K; i++)
    {
        cout << h_C[i] << "  ";
        if ((i+1)%K==0)
            cout << endl;
    }
	
	// 销毁句柄
	hipblasDestroy(handle);
	
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
	
    return 0;
}